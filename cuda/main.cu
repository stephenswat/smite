#include "hip/hip_runtime.h"
#include <random>
#include <sstream>
#include <iostream>
#include <fstream>
#include <iomanip>

#include <hip/hip_runtime.h>

#include <boost/program_options.hpp>
#include <boost/log/trivial.hpp>

#define cudaErrorCheck(r) { _cudaErrorCheck((r), __FILE__, __LINE__); }

void _cudaErrorCheck(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) {
        std::stringstream ss;

        ss << "[" << file << ":" << line << "] CUDA error: " << hipGetErrorString(code);

        throw std::runtime_error(ss.str());
    }
}

static constexpr int WARP_SIZE = 32;
static constexpr int MAT_DIM = 8;

/**
 * Calculate a = a * b;
 */
__device__ void mat_mul(float * a, float * b) {
    float tmp[MAT_DIM * MAT_DIM];

    for (uint i = 0; i < MAT_DIM; ++i) {
        for (uint j = 0; j < MAT_DIM; ++j) {
            tmp[i * MAT_DIM + j] = a[i * MAT_DIM + j];
        }
    }

    for (uint i = 0; i < MAT_DIM * MAT_DIM; ++i) {
        a[i] = 0.f;
    }

    for (uint i = 0; i < MAT_DIM; ++i) {
        for (uint j = 0; j < MAT_DIM; ++j) {
            for (uint k = 0; k < MAT_DIM; ++k) {
                a[MAT_DIM * i + j] += tmp[MAT_DIM * i + k] * b[MAT_DIM * k + j];
            }
        }
    }
}

template<bool Sync>
__global__ void kernel(uint tc, float * ms, float * os, uint * ps, uint * cs1, uint * cs2) {
    int gid = blockIdx.x;
    int tid = threadIdx.x + blockIdx.x * tc;
    int lid = threadIdx.x;

    if (lid >= tc) {
        return;
    }

    float acc[MAT_DIM * MAT_DIM];

    for (uint i = 0; i < MAT_DIM; ++i) {
        for (uint j = 0; j < MAT_DIM; ++j) {
            acc[i * MAT_DIM + j] = i == j ? 1.f : 0.f;
        }
    }

    float mat[MAT_DIM * MAT_DIM];

    for (uint i = 0; i < MAT_DIM * MAT_DIM; ++i) {
        mat[i] = ms[MAT_DIM * MAT_DIM * tid + i];
    }

    __syncthreads();

    uint t_start = clock64();
    uint t_mimt = clock64();
    
    if constexpr (Sync) {
        __syncwarp();
    }

    for (uint i = 0; i < ps[tid]; ++i) {
        mat_mul(acc, mat);
        if constexpr (Sync) {
            __syncwarp(__activemask());
        }
        t_mimt = clock64();
    }

    if constexpr (Sync) {
        __syncwarp();
    }

    uint t_simt = clock64();

    __syncthreads();

    for (uint i = 0; i < MAT_DIM * MAT_DIM; ++i) {
        os[MAT_DIM * MAT_DIM * tid + i] = acc[i];
    }

    __syncthreads();

    if (lid == 0) {
        cs1[gid] = 0;
        cs2[gid] = 0;
    }

    __syncthreads();

    atomicAdd(&cs1[gid], t_simt - t_start);
    atomicAdd(&cs2[gid], t_mimt - t_start);
}

template<template <typename> typename RNG, typename ...Args>
void generate_ps(uint * o, uint n, std::mt19937 & r, Args... args) {
    RNG<uint> pdis(std::forward<Args>(args)...);

    for (uint i = 0; i < n; ++i) {
        o[i] = pdis(r);
    }
}

void parse_opts(int argc, char* argv[], boost::program_options::variables_map & vm) {
    boost::program_options::options_description opts("general options");

    opts.add_options()
        (
            "help",
            "produce help message"
        )
        (
            "distribution,d",
            boost::program_options::value<std::string>()->required(),
            "distribution type to use (must be \"uniform\", \"binomial\", \"nbinomial\", \"poisson\", or \"geometric\")"
        )
        (
            "seed",
            boost::program_options::value<std::mt19937::result_type>(),
            "set seed for random number generators"
        )
        (
            "output,o",
            boost::program_options::value<std::string>(),
            "file name to write results to"
        )
        (
            "threads,t",
            boost::program_options::value<uint>()->default_value(32),
            "number of threads to run in lock-step"
        )
        (
            "samples,s",
            boost::program_options::value<uint>()->default_value(131072),
            "number of samples to run"
        )
        (
            "smem",
            boost::program_options::value<uint>(),
            "bytes of shared memory to occupy per warp"
        )
        (
            "sync",
            "enable explicit warp synchronization"
        )
    ;

    boost::program_options::options_description opts_dist("distribution options");

    opts_dist.add_options()
        (
            "low",
            boost::program_options::value<uint>(),
            "lower bound for support (inclusive) (uniform)"
        )
        (
            "high",
            boost::program_options::value<uint>(),
            "upper bound for support (inclusive) (uniform)"
        )
        (
            "lambda",
            boost::program_options::value<float>(),
            "rate of events (poisson)"
        )
        (
            "probability",
            boost::program_options::value<float>(),
            "rate of events (binomial, geometric)"
        )
        (
            "trials",
            boost::program_options::value<uint>(),
            "rate of events (binomial)"
        )
        (
            "failures",
            boost::program_options::value<uint>(),
            "failure limit (nbinomial)"
        )
    ;

    opts.add(opts_dist);

    boost::program_options::options_description opts_dist_poisson("poisson distribution options");

    opts_dist_poisson.add_options()
    ;

    opts.add(opts_dist_poisson);

    boost::program_options::parsed_options parsed = boost::program_options::command_line_parser(
        argc, argv
    )
    .options(opts)
    .allow_unregistered()
    .run();

    boost::program_options::store(parsed, vm);

    if (vm.count("help")) {
        std::cout << opts << std::endl;
        std::exit(0);
    }

    try {
        boost::program_options::notify(vm);
    } catch (boost::program_options::required_option & e) {
        BOOST_LOG_TRIVIAL(error) << e.what();
        std::exit(1);
    }

    std::string dist = vm["distribution"].as<std::string>();

    if (dist == "uniform") {
        if (!vm.count("low")) {
            BOOST_LOG_TRIVIAL(error) << "Uniform distribution requires \"low\" to be specified";
            std::exit(1);
        }
        if (!vm.count("high")) {
            BOOST_LOG_TRIVIAL(error) << "Uniform distribution requires \"high\" to be specified";
            std::exit(1);
        }
    } else if (dist == "poisson") {
        if (!vm.count("lambda")) {
            BOOST_LOG_TRIVIAL(error) << "Poisson distribution requires \"lambda\" to be specified";
            std::exit(1);
        }
    } else if (dist == "geometric") {
        if (!vm.count("probability")) {
            BOOST_LOG_TRIVIAL(error) << "Geometric distribution requires \"probability\" to be specified";
            std::exit(1);
        }
    } else if (dist == "binomial") {
        if (!vm.count("probability")) {
            BOOST_LOG_TRIVIAL(error) << "Binomial distribution requires \"probability\" to be specified";
            std::exit(1);
        }
        if (!vm.count("trials")) {
            BOOST_LOG_TRIVIAL(error) << "Binomial distribution requires \"trials\" to be specified";
            std::exit(1);
        }
    } else if (dist == "nbinomial") {
        if (!vm.count("probability")) {
            BOOST_LOG_TRIVIAL(error) << "Negative binomial distribution requires \"probability\" to be specified";
            std::exit(1);
        }
        if (!vm.count("failures")) {
            BOOST_LOG_TRIVIAL(error) << "Negative binomial distribution requires \"failures\" to be specified";
            std::exit(1);
        }
    } else {
        BOOST_LOG_TRIVIAL(error) << "Invalid distribution \"" << dist << "\"";
        std::exit(1);
    }
}

int main(int argc, char* argv[]) {
    boost::program_options::variables_map vm;

    parse_opts(argc, argv, vm);

    std::mt19937::result_type seed;

    if (vm.count("seed")) {
        BOOST_LOG_TRIVIAL(info) << "Seeding randon number generator with user-provided seed";
        seed = vm["seed"].as<std::mt19937::result_type>();
    } else {
        BOOST_LOG_TRIVIAL(info) << "Seeding randon number generator with system-provided seed";
        std::random_device rd;
        seed = rd();
    }

    BOOST_LOG_TRIVIAL(info) << "Initializing Mersenne twister with seed " << seed;

    // std::random_device rd;
    std::mt19937 gen(seed);

    uint group_size = vm["threads"].as<uint>();
    uint group_count = vm["samples"].as<uint>();

    BOOST_LOG_TRIVIAL(info) << "Setting work group size to " << group_size;
    BOOST_LOG_TRIVIAL(info) << "Setting work set count to " << group_count;

    uint work_items = group_size * group_count;

    BOOST_LOG_TRIVIAL(info) << "Total number of work items is " << work_items;

    /*
    * Prepare the array of matrices.
    */
    BOOST_LOG_TRIVIAL(info) << "Preparing input matrices of size " << MAT_DIM << "x" << MAT_DIM;

    std::uniform_real_distribution<float> mdis(0.0f, 1.0f);
    float * _ms = new float[MAT_DIM * MAT_DIM * work_items];

    for (std::size_t i = 0; i < MAT_DIM * MAT_DIM * work_items; ++i) {
        _ms[i] = mdis(gen);
    }

    std::size_t input_matrix_bytes = MAT_DIM * MAT_DIM * work_items * sizeof(float);
    
    BOOST_LOG_TRIVIAL(info) << "Transfering input matrices to device (total size " << input_matrix_bytes << " bytes)";

    float * ms;

    cudaErrorCheck(hipMalloc(&ms, input_matrix_bytes));
    cudaErrorCheck(hipMemcpy(ms, _ms, input_matrix_bytes, hipMemcpyHostToDevice));

    /*
     * Prepare the array of powers to which we raise our matrices.
     */
    BOOST_LOG_TRIVIAL(info) << "Preparing number of iterations per work item";

    uint * _ps = new uint[work_items];

    std::string dist = vm["distribution"].as<std::string>();

    if (dist == "uniform") {
        generate_ps<std::uniform_int_distribution>(_ps, work_items, gen, vm["low"].as<uint>(), vm["high"].as<uint>());
    } else if (dist == "poisson") {
        generate_ps<std::poisson_distribution>(_ps, work_items, gen, vm["lambda"].as<float>());
    } else if (dist == "geometric") {
        generate_ps<std::geometric_distribution>(_ps, work_items, gen, vm["probability"].as<float>());

        for (std::size_t i = 0; i < work_items; ++i) {
            _ps[i]++;
        }
    } else if (dist == "binomial") {
        generate_ps<std::binomial_distribution>(_ps, work_items, gen, vm["trials"].as<uint>(), vm["probability"].as<float>());
    } else if (dist == "nbinomial") {
        generate_ps<std::negative_binomial_distribution>(_ps, work_items, gen, vm["failures"].as<uint>(), vm["probability"].as<float>());
    }

    BOOST_LOG_TRIVIAL(info) << "Transfering iteration counts to device (total size " << (work_items * sizeof(uint)) << " bytes)";

    uint * ps;

    cudaErrorCheck(hipMalloc(&ps, work_items * sizeof(uint)));
    cudaErrorCheck(hipMemcpy(ps, _ps, work_items * sizeof(uint), hipMemcpyHostToDevice));

    /*
     * Prepare the output arrays on the device.
     */
    float * os;
    uint * cs_simt, * cs_mimt;

    BOOST_LOG_TRIVIAL(info) << "Allocating memory for output matrices (total size " << (MAT_DIM * MAT_DIM * work_items * sizeof(float)) << " bytes)";

    cudaErrorCheck(hipMalloc(&os, MAT_DIM * MAT_DIM * work_items * sizeof(float)));

    BOOST_LOG_TRIVIAL(info) << "Allocating memory for timing information (total size " << (2 * group_count * sizeof(uint)) << " bytes)";

    cudaErrorCheck(hipMalloc(&cs_simt, group_count * sizeof(uint)));
    cudaErrorCheck(hipMalloc(&cs_mimt, group_count * sizeof(uint)));

    /*
     * Prepare the output arrays on the host.
     */
    BOOST_LOG_TRIVIAL(info) << "Allocating host memory for output data";

    uint * _cs_simt = new uint[work_items];
    uint * _cs_mimt = new uint[work_items];

    /*
     * Run the SIMT simulation.
     */
    cudaErrorCheck(hipDeviceSynchronize());

    int dev_id;
    hipDeviceProp_t props;

    cudaErrorCheck(hipGetDevice(&dev_id));
    cudaErrorCheck(hipGetDeviceProperties(&props, dev_id));

    BOOST_LOG_TRIVIAL(info) << "Device info:";
    BOOST_LOG_TRIVIAL(info) << "    Name: " << props.name;
    BOOST_LOG_TRIVIAL(info) << "    Max shared memory per block: " << props.sharedMemPerBlockOptin << "B";
    BOOST_LOG_TRIVIAL(info) << "    CC version: " << props.major << "." << props.minor;

    uint smem_target;

    if (vm.count("smem")) {
        smem_target = vm["smem"].as<uint>();
    } else {
        smem_target = props.sharedMemPerBlockOptin;
    }

    if (smem_target > props.sharedMemPerBlock) {
        BOOST_LOG_TRIVIAL(info) << "Setting maximum shared memory to " << props.sharedMemPerBlockOptin << "B";

        cudaErrorCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel<true>), hipFuncAttributeMaxDynamicSharedMemorySize, props.sharedMemPerBlockOptin));
        cudaErrorCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel<false>), hipFuncAttributeMaxDynamicSharedMemorySize, props.sharedMemPerBlockOptin));
    }

    BOOST_LOG_TRIVIAL(info) << "Invoking kernel:";
    BOOST_LOG_TRIVIAL(info) << "    " << group_count << " blocks";
    BOOST_LOG_TRIVIAL(info) << "    " << WARP_SIZE << " threads per block";
    BOOST_LOG_TRIVIAL(info) << "    " << smem_target << " bytes of shared memory per block";

    if (vm.count("sync")) {
        BOOST_LOG_TRIVIAL(info) << "    Running in synchronized mode";
        kernel<true><<<group_count, WARP_SIZE, smem_target>>>(group_size, ms, os, ps, cs_simt, cs_mimt);
    } else {
        BOOST_LOG_TRIVIAL(info) << "    Running in unsynchronized mode";
        kernel<false><<<group_count, WARP_SIZE, smem_target>>>(group_size, ms, os, ps, cs_simt, cs_mimt);
    }
    
    cudaErrorCheck(hipPeekAtLastError());

    BOOST_LOG_TRIVIAL(info) << "Awaiting kernel synchronization";

    cudaErrorCheck(hipDeviceSynchronize());

    BOOST_LOG_TRIVIAL(info) << "Transfering results from device to host";

    cudaErrorCheck(hipMemcpy(_cs_simt, cs_simt, group_count * sizeof(uint), hipMemcpyDeviceToHost));
    cudaErrorCheck(hipMemcpy(_cs_mimt, cs_mimt, group_count * sizeof(uint), hipMemcpyDeviceToHost));

    cudaErrorCheck(hipDeviceSynchronize());

    BOOST_LOG_TRIVIAL(info) << "Deallocating device memory";

    cudaErrorCheck(hipFree(ms));
    cudaErrorCheck(hipFree(ps));
    cudaErrorCheck(hipFree(os));
    cudaErrorCheck(hipFree(cs_simt));
    cudaErrorCheck(hipFree(cs_mimt));

    BOOST_LOG_TRIVIAL(info) << "Results succesfully validated";

    std::string ofilename;

    if (vm.count("output")) {
        ofilename = vm["output"].as<std::string>();
    } else {
        std::stringstream ss;

        ss << "data_";

        if (dist == "uniform") {
            ss << "uniform_" << vm["low"].as<uint>() << "_" << vm["high"].as<uint>();
        } else if (dist == "poisson") {
            ss << "pois_" << vm["lambda"].as<float>();
        } else if (dist == "geometric") {
            ss << "geo_" << std::setfill('0') << std::setw(3) << static_cast<int>(100 * vm["probability"].as<float>());
        } else if (dist == "binomial") {
            ss << "binom_" << vm["trials"].as<uint>() << "_" << std::setfill('0') << std::setw(3) << static_cast<int>(100 * vm["probability"].as<float>());
        } else if (dist == "nbinomial") {
            ss << "nbinom_" << vm["failures"].as<uint>() << "_" << std::setfill('0') << std::setw(3) << static_cast<int>(100 * vm["probability"].as<float>());
        }

        ss << "_" << group_size << ".csv";

        ofilename = ss.str();
    }

    BOOST_LOG_TRIVIAL(info) << "Writing results to file " << ofilename;

    std::ofstream ofile;
    ofile.open(ofilename);

    ofile << "i,sim_simt,sim_mimt,mea_simt,mea_mimt" << std::endl;

    for (uint i = 0; i < group_count; ++i) {
        uint sum = 0;
        uint max = 0;

        for (uint j = 0; j < group_size; ++j) {
            max = std::max(max, _ps[i * group_size + j]);
            sum += _ps[i * group_size + j];
        }

        ofile << i << "," << (group_size * max) << "," << sum << "," << _cs_simt[i] << "," << _cs_mimt[i] << std::endl;
    }
    ofile.close();

    BOOST_LOG_TRIVIAL(info) << "Run complete, goodbye!";

    return 0;
}
